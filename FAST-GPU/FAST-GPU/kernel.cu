#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "opencv2/imgcodecs/imgcodecs.hpp"
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <time.h>

#define PADDING 3
#define BLOCK_SIZE 32  // max 32
#define CIRCLE_SIZE 16
#define PI 12		   // contiguous pixels
#define THRESHOLD 75
#define MASK_SIZE 7	   // Non-maximal suppression (must be odd nummber here), when MASK_SIZE > 7 it access memory outside the image
//#define USE_SHARED

// host
unsigned char *h_img;
unsigned short *h_candidates;
int *h_circle;
int *h_mask;
// time
clock_t start, end;
double time_measured;

// device
unsigned char *d_img;
unsigned short *d_candidates;
__constant__ int d_circle[CIRCLE_SIZE];
__constant__ int d_mask[MASK_SIZE*MASK_SIZE];


static void HandleError(hipError_t error, const char *file, int line) {
	if (error != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(error), file, line);
		scanf(" ");
		exit(EXIT_FAILURE);
	}
}

#define CHECK_ERROR( error ) ( HandleError( error, __FILE__, __LINE__ ) )

__host__ void create_circle(int *circle, int w) {
	// create surrounding circle using given width
	circle[0] = -3*w;
	circle[1] = -3*w + 1;
	circle[2] = -2*w + 2;
	circle[3] = -w + 3;
	
	circle[4] = 3;
	circle[5] = w + 3;
	circle[6] = 2*w + 2;
	circle[7] = 3*w + 1;

	circle[8] = 3*w;
	circle[9] = 3*w - 1;
	circle[10] = 2*w - 2;
	circle[11] = w - 3;

	circle[12] = -3;
	circle[13] = -w - 3;
	circle[14] = -2*w - 2;
	circle[15] = -3*w - 1;
}

__host__ void create_mask(int *mask, int w) {
	// create mask with given defined mask size and width
	int start = (int)-MASK_SIZE / 2;
	int end = (int)MASK_SIZE / 2;
	int index = 0;
	for (int i = start; i <= end; i++)
	{
		for (int j = start; j <= end; j++)
		{
			mask[index] = i * w + j;
			index++;
		}
	}
}

__device__ char comparator(unsigned char pixel_val, unsigned char circle_val) {
	// very similar to get_score, only returns normalised values
	if (circle_val > (pixel_val + THRESHOLD)) {
		return 1;
	}
	else {
		if (circle_val < (pixel_val - THRESHOLD)) {
			return -1;
		}
		else {
			return 0;
		}
	}
}

__device__ char get_score(unsigned char pixel_val, unsigned char circle_val) {
	// returns circle element score, positive when higher, negative when lower intensity
	char val = pixel_val + THRESHOLD;
	if (circle_val > val) {
		return circle_val - val;
	}
	else {
		val = pixel_val - THRESHOLD;
		if (circle_val < val) {
			return -(val - circle_val);
		}
		else {
			return 0;
		}
	}
}

__device__ int coords_2to1(int x, int y, int width, int height, bool eliminate_padding) {
	// recalculate 2d indexes into 1d array
	if (eliminate_padding && ((x - PADDING) < 0 || (x + PADDING) >= width || (y - PADDING) < 0 || (y + PADDING) >= height)) {
		// cutout the borders of image
		return -1;
	}
	else {
		return x + y * width;
	}
}

__global__ void FAST_shared(unsigned char *input, unsigned short *output, int width, int height)
{
	extern __shared__ unsigned char sData[];
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	// get 1d coordinates and cutout borders
	int id1d = coords_2to1(idx, idy, width, height, true);
	if (id1d == -1) {
		return;
	}
	// fill in shared memory
	int shared_width = BLOCK_SIZE + (2*PADDING);
	int s_mem_half_size = ((shared_width)*(shared_width))/2;
	int index1 = coords_2to1(threadIdx.x, threadIdx.y, shared_width, shared_width, false);
	if (index1 < s_mem_half_size) {
		int index2 = index1 + s_mem_half_size;
		int global_x1 = -PADDING + (index1 % shared_width) + blockIdx.x * blockDim.x;
		int global_y1 = -PADDING + (index1 / shared_width) + blockIdx.y * blockDim.y;
		int global_x2 = -PADDING + (index2 % shared_width) + blockIdx.x * blockDim.x;
		int global_y2 = -PADDING + (index2 / shared_width) + blockIdx.y * blockDim.y;
		sData[index1] = input[coords_2to1(global_x1, global_y1, width, height, false)];
		sData[index2] = input[coords_2to1(global_x2, global_y2, width, height, false)];
	}
	// fast test
	int s_id1d = coords_2to1(threadIdx.x + PADDING, threadIdx.y + PADDING, shared_width, shared_width, false);
	unsigned char pixel = sData[s_id1d];
	char top = comparator(pixel, sData[s_id1d + d_circle[0]]);
	char down = comparator(pixel, sData[s_id1d + d_circle[8]]);
	char right = comparator(pixel, sData[s_id1d + d_circle[4]]);
	char left = comparator(pixel, sData[s_id1d + d_circle[12]]);
	if (abs(top + down + right + left) < 2 || (abs(top + down) < 2 && abs(left + right) < 2)) {
		return;
	}
	// make complex test and calculate score
	char score;
	int score_sum = 0;
	int max_score = 0;
	char val;
	char last_val = -2;
	unsigned char consecutive = 0;
	bool corner = false;
	for (size_t i = 0; i < (CIRCLE_SIZE + PI); i++) // iterate over whole circle
	{
		if (consecutive >= 12) {
			corner = true;
		}
		score = get_score(pixel, sData[s_id1d + d_circle[i % CIRCLE_SIZE]]);
		val = (score < 0) ? -1 : (score > 0);  // signum
		if (val == last_val) {
			consecutive++;
			score_sum += abs(score);
		}
		else {
			if (score_sum > max_score) {
				max_score = score_sum;
			}
			consecutive = 1;
			score_sum = abs(score);
		}
		last_val = val;
	}
	if (score_sum > max_score) {
		max_score = score_sum;
	}
	if (corner) {
		output[id1d] = (unsigned short) max_score;
	}
	else {
		return;
	}
	__syncthreads();
	// non-maximal suppresion (very time consuming)
	for (size_t i = 0; i < MASK_SIZE*MASK_SIZE; i++)
	{
		if (output[id1d + d_mask[i]] > max_score) {
			return;
		}
	}
	for (size_t i = 0; i < MASK_SIZE*MASK_SIZE; i++)	// if this thread has max value on id1d delete everything around in filter
	{
		if (d_mask[i]) {
			output[id1d + d_mask[i]] = 0;
		}
	}
	return;
}

__global__ void FAST_global(unsigned char *input, unsigned short *output, int width, int height)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx == 0 && idy == 0) {
		for (size_t i = 0; i < MASK_SIZE*MASK_SIZE; i++)
		{
			printf("mask: %d\n", d_mask[i]);
		}
	}
	// get 1d coordinates and cutout borders
	int id1d = coords_2to1(idx, idy, width, height, true);
	if (id1d == -1) {
		return;
	}
	// fast test
	unsigned char pixel = input[id1d];
	char top = comparator(pixel, input[id1d + d_circle[0]]);
	char down = comparator(pixel, input[id1d + d_circle[8]]);
	char right = comparator(pixel, input[id1d + d_circle[4]]);
	char left = comparator(pixel, input[id1d + d_circle[12]]);
	if (abs(top + down + right + left) < 2 || (abs(top + down) < 2 && abs(left + right) < 2)) {
		return;
	}
	// make complex test and calculate score
	char score;
	int score_sum = 0;
	int max_score = 0;
	char val;
	char last_val = -2;
	unsigned char consecutive = 0;
	bool corner = false;
	for (size_t i = 0; i < (CIRCLE_SIZE+PI); i++) // iterate over whole circle
	{
		if (consecutive >= 12) {
			corner = true;
		}
		score = get_score(pixel, input[id1d + d_circle[i % CIRCLE_SIZE]]);
		val = (score < 0) ? -1 : (score > 0);  // signum
		if (val == last_val) {
			consecutive++;
			score_sum += abs(score);
		}
		else {
			if (score_sum > max_score) {
				max_score = score_sum;
			}
			consecutive = 1;
			score_sum = abs(score);
		}
		last_val = val;
	}
	if (score_sum > max_score) {
		max_score = score_sum;
	}
	if (corner) {
		output[id1d] = (unsigned short) max_score;
	}
	else {
		return;
	}
	__syncthreads();
	printf("MADE IT HERE!\n");
	// non-maximal suppresion (very time consuming)
	for (size_t i = 0; i < MASK_SIZE*MASK_SIZE; i++)
	{
		if (output[id1d + d_mask[i]] > max_score) {
			return;
		}
	}
	for (size_t i = 0; i < MASK_SIZE*MASK_SIZE; i++)	// if this thread has max value on id1d delete everything around in filter
	{
		if (d_mask[i]) {
			output[id1d + d_mask[i]] = 0;
		}
	}
	return;
}

void show_image(cv::Mat img) {
	cv::namedWindow("Display window", cv::WINDOW_AUTOSIZE); // Create a window for display.
	//cv::Size size(140, 100);
	//cv::resize(img, img, size);
	cv::imshow("Display window", img);
	cv::waitKey(0);
}

int main(int argc, char **argv)
{
	// load image
	cv::Mat image;
	image = cv::imread("..\\..\\cvut.png", 0);

	// resize image for testing small image
	cv::Size size(768, 1024);
	resize(image, image, size);

	// get dimension of image
	int width = image.cols;
	int height = image.rows;
	int length = width * height;
	int shared_width = BLOCK_SIZE + (2 * PADDING);
	size_t char_size = length * sizeof(unsigned char);
	size_t short_size = length * sizeof(unsigned short);
	printf("\n --- Image loaded --- \n");

	// allocate memory
	h_img = (unsigned char*)malloc(char_size);
	h_candidates = (unsigned short*)malloc(short_size);
	h_circle = (int*)malloc(CIRCLE_SIZE*sizeof(int));
	h_mask = (int*)malloc(MASK_SIZE*MASK_SIZE*sizeof(int));
	CHECK_ERROR(hipMalloc((void**)&d_img, char_size));
	CHECK_ERROR(hipMalloc((void**)&d_candidates, short_size));
	CHECK_ERROR(hipMemset(d_candidates, 0, short_size));

	// create array from image
	for (int i = 0; i < length; i++)
	{
		h_img[i] = image.at<unsigned char>((int)i / image.cols, i % image.cols);
	}

	// create circle and copy to device
	#ifdef USE_SHARED
	create_circle(h_circle, shared_width);
	create_mask(h_mask, width);
	#else
	create_circle(h_circle, width);
	create_mask(h_mask, width);
	#endif
	CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_circle), h_circle, CIRCLE_SIZE * sizeof(int)));
	CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_mask), h_mask, MASK_SIZE * MASK_SIZE * sizeof(int)));

	// copy image to device
	CHECK_ERROR(hipMemcpy(d_img, h_img, char_size, hipMemcpyHostToDevice));

	// define grid and block sizes
	dim3 blocks(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(((int) (width-1) / BLOCK_SIZE) + 1, ((int) (height-1) / BLOCK_SIZE) + 1);

	// run kernel and measure the time
	printf(" --- Memory allocated, running kernel --- \n");
	start = clock();
	#ifdef USE_SHARED
	FAST_shared <<< grid, blocks, shared_width*shared_width*sizeof(unsigned char) >>> (d_img, d_candidates, image.cols, image.rows);
	#else
	FAST_global <<< grid, blocks >>> (d_img, d_candidates, image.cols, image.rows);
	#endif
	CHECK_ERROR(hipDeviceSynchronize());
	end = clock();
	time_measured = ((double)(end - start)) / CLOCKS_PER_SEC;
	printf(" --- Image with size (%d, %d) was processed in %f sec --- \n", width, height, time_measured);

	// copy result to host
	CHECK_ERROR(hipMemcpy(h_candidates, d_candidates, short_size, hipMemcpyDeviceToHost));

	printf(" --- Result copied from device to host --- \n");
	// draw corners 
	cv::cvtColor(image, image, cv::COLOR_GRAY2BGR);
	for (int i = 0; i < width; i++)
	{
		for (int j = 0; j < height; j++)
		{
			if (h_candidates[i + j * width]) {
				cv::circle(image, cv::Point(i, j), 3, cv::Scalar(0, 255, 0));
			}
		}
	}

	// show image
	show_image(image);

	// free all memory
	CHECK_ERROR(hipFree(d_img));
	CHECK_ERROR(hipFree(d_candidates));
	free(h_img);
	free(h_candidates);
	free(h_mask);
	free(h_circle);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
	CHECK_ERROR(hipDeviceReset());

    return 0;
}
